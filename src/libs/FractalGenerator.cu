#include "hip/hip_runtime.h"
#include "FractalGenerator.cuh"

#include <iostream>

#include <opencv2/opencv.hpp>


namespace Fractal
{
    static constexpr auto CHANNELS{3U};

    static constexpr dim3 BLOCK_SIZE{32U, 32U};

    static constexpr float MAX_COLOR{255.0F};

    static Size imageSize{0U, 0U};

    static bool isRendered{false};

    static std::uint8_t * deviceImage{nullptr};
    static std::vector<std::uint8_t> hostImage;


    __host__ __forceinline__ static auto CleanUp() -> void;

    __device__ __forceinline__ static auto PixelToPoint(Pixel pixel, Size size, Point topLeft, Point bottomRight) -> Point;


    __global__ static auto MandelbrotKernel(std::uint8_t * image, Size size, std::size_t iterations, float logIterations, Point topLeft, Point bottomRight, float radiusSquared) -> void;

    __device__ __forceinline__ static auto GenerateMandelbrotPixel(Point startPoint, std::size_t iterations, float logIterations, float radiusSquared) -> std::uint8_t;


    __global__ static auto JuliaKernel(std::uint8_t * image, Size size, std::size_t iterations, float logIterations, Point topLeft, Point bottomRight, float radiusSquared) -> void;

    __device__ __forceinline__ static auto GenerateJuliaPixel(Point startPoint, std::size_t iterations, float logIterations, float radiusSquared) -> std::uint8_t;


    __global__ static auto TricornKernel(std::uint8_t * image, Size size, std::size_t iterations, float logIterations, Point topLeft, Point bottomRight, float radiusSquared) -> void;

    __device__ __forceinline__ static auto GenerateTricornPixel(Point startPoint, std::size_t iterations, float logIterations, float radiusSquared) -> std::uint8_t;


    __global__ static auto CosineKernel(std::uint8_t * image, Size size, std::size_t iterations, float logIterations, Point topLeft, Point bottomRight, float radiusSquared) -> void;

    __device__ __forceinline__ static auto GenerateCosinePixel(Point startPoint, std::size_t iterations, float logIterations, float radiusSquared) -> std::uint8_t;


    __host__ auto GeneratorConstruct(Size const & size) -> void
    {
        if (auto const result{hipMalloc(&deviceImage, size.width * size.height * CHANNELS * sizeof(std::uint8_t))}; hipSuccess != result)
        {
            std::cerr << "Error allocating memory for Mandelbrot image: " << hipGetErrorString(result) << '\n';
            std::exit(EXIT_FAILURE);
        }

        hostImage.resize(size.width * size.height * CHANNELS);

        imageSize = size;
    }

    __host__ auto GeneratorDestruct() -> void
    {
        hipFree(deviceImage);
        deviceImage = nullptr;

        hostImage.clear();

        isRendered = false;

        imageSize = {0U, 0U};

        hipDeviceReset();
    }

    __host__ auto Render(Type const type, Point const topLeft, Point const bottomRight, float const radiusSquared, std::size_t const iterations) -> void
    {
        static const dim3 GRID_SIZE{
            static_cast<uint32_t>((imageSize.width + BLOCK_SIZE.x - 1U) / BLOCK_SIZE.x),
            static_cast<uint32_t>((imageSize.height + BLOCK_SIZE.y - 1U) / BLOCK_SIZE.y)
        };

        auto const logMaxIterations{std::log(static_cast<float>(iterations))};

        switch (type)
        {
            case Type::MANDELBROT :
                MandelbrotKernel<<<GRID_SIZE, BLOCK_SIZE>>>(deviceImage, imageSize, iterations, logMaxIterations, topLeft, bottomRight, radiusSquared);
                break;
            case Type::JULIA :
                JuliaKernel<<<GRID_SIZE, BLOCK_SIZE>>>(deviceImage, imageSize, iterations, logMaxIterations, topLeft, bottomRight, radiusSquared);
                break;
            case Type::TRICORN :
                TricornKernel<<<GRID_SIZE, BLOCK_SIZE>>>(deviceImage, imageSize, iterations, logMaxIterations, topLeft, bottomRight, radiusSquared);
                break;
            case Type::COSINE :
                CosineKernel<<<GRID_SIZE, BLOCK_SIZE>>>(deviceImage, imageSize, iterations, logMaxIterations, topLeft, bottomRight, radiusSquared);
                break;
            default :
                std::cerr << "Unknown fractal type.\n";
                CleanUp();
        }

        hipDeviceSynchronize();

        if (auto const error{hipGetLastError()}; hipSuccess != error)
        {
            std::cerr << "Error in the " << TypeToString(type) << " kernel: " << hipGetErrorString(error) << '\n';
            CleanUp();
        }
    }

    __host__ auto Retrieve() -> void
    {
        hipMemcpy(hostImage.data(), deviceImage, hostImage.size() * sizeof(std::uint8_t), hipMemcpyDeviceToHost);

        isRendered = true;
    }

    __host__ auto GetImage() -> std::uint8_t const *
    {
        if (!isRendered)
        {
            throw std::runtime_error("The fractal has not been rendered yet.");
        }

        return hostImage.data();
    }

    __host__ auto Save(std::string_view const & filename) -> void
    {
        using namespace cv;

        if (!isRendered)
        {
            throw std::runtime_error("The fractal has not been rendered yet.");
        }

        Mat const greyImage(static_cast<int>(imageSize.height), static_cast<int>(imageSize.width), CV_8UC1, hostImage.data());
        Mat coloredImage;
        applyColorMap(greyImage, coloredImage, COLORMAP_MAGMA);
        imwrite(filename.data(), coloredImage);
    }

    __host__ __forceinline__ static auto CleanUp() -> void
    {
        GeneratorDestruct();
        std::exit(EXIT_FAILURE);
    }

    auto TypeToString(Type const type) -> std::string_view
    {
        switch (type)
        {
            case MANDELBROT :
                return "Mandelbrot";
            case JULIA :
                return "Julia";
            case TRICORN :
                return "Tricorn";
            case COSINE :
                return "Cosine";
            default :
                throw std::runtime_error("Unknown fractal type.");
        }
    }

    __device__ auto PixelToPoint(Pixel const pixel, Size const size, Point const topLeft, Point const bottomRight) -> Point
    {
        auto const domainWidth{bottomRight.real - topLeft.real};
        auto const domainHeight{topLeft.imag - bottomRight.imag};

        auto const real{topLeft.real + (static_cast<decltype(domainWidth)>(pixel.x) * domainWidth / static_cast<decltype(domainWidth)>(size.width))};
        auto const imag{topLeft.imag - (static_cast<decltype(domainHeight)>(pixel.y) * domainHeight / static_cast<decltype(domainHeight)>(size.height))};

        return {real, imag};
    }

    __global__ static auto MandelbrotKernel(std::uint8_t * const image, Size const size, std::size_t const iterations, float const logIterations, Point const topLeft,
    Point const bottomRight, float const radiusSquared) -> void
    {
        auto const row{(blockIdx.y * blockDim.y) + threadIdx.y};
        auto const col{(blockIdx.x * blockDim.x) + threadIdx.x};

        if (row < size.height && col < size.width)
        {
            auto const point = PixelToPoint({col, row}, size, topLeft, bottomRight);
            auto const value = GenerateMandelbrotPixel(point, iterations, logIterations, radiusSquared);
            image[(row * size.width + col) * CHANNELS + 0] = value;
            image[(row * size.width + col) * CHANNELS + 1] = value;
            image[(row * size.width + col) * CHANNELS + 2] = value;
        }
    }

    __device__ auto GenerateMandelbrotPixel(Point const startPoint, std::size_t const iterations, float const logIterations, float const radiusSquared) -> std::uint8_t
    {
        Point point{0.0, 0.0};

        for (std::size_t iteration = 0; iteration < iterations; ++iteration)
        {
            if (point.norm() > radiusSquared)
            {
                return static_cast<std::uint8_t>(MAX_COLOR * logf(iteration + 1) / logIterations);
            }

            point = point * point + startPoint;
        }

        return 0;
    }

    __global__ static auto JuliaKernel(std::uint8_t * const image, Size const size, std::size_t const iterations, float const logIterations, Point const topLeft,
    Point const bottomRight, float const radiusSquared) -> void
    {
        auto const row{(blockIdx.y * blockDim.y) + threadIdx.y};
        auto const col{(blockIdx.x * blockDim.x) + threadIdx.x};

        if (row < size.height && col < size.width)
        {
            auto const point = PixelToPoint({col, row}, size, topLeft, bottomRight);
            auto const value = GenerateJuliaPixel(point, iterations, logIterations, radiusSquared);
            image[(row * size.width + col) * CHANNELS + 0] = value;
            image[(row * size.width + col) * CHANNELS + 1] = value;
            image[(row * size.width + col) * CHANNELS + 2] = value;
        }
    }

    __device__ auto GenerateJuliaPixel(Point const startPoint, std::size_t const iterations, float const logIterations, float const radiusSquared) -> std::uint8_t
    {
        __device__ static constexpr Point C_POINT{-0.7, 0.27015};

        Point point{startPoint};

        for (std::size_t iteration = 0; iteration < iterations; ++iteration)
        {
            if (point.norm() > radiusSquared)
            {
                return static_cast<std::uint8_t>(MAX_COLOR * logf(iteration + 1) / logIterations);
            }

            point = point * point + C_POINT;
        }

        return 0;
    }

    __global__ static auto TricornKernel(std::uint8_t * const image, Size const size, std::size_t const iterations, float const logIterations, Point const topLeft,
    Point const bottomRight, float const radiusSquared) -> void
    {
        auto const row{(blockIdx.y * blockDim.y) + threadIdx.y};
        auto const col{(blockIdx.x * blockDim.x) + threadIdx.x};

        if (row < size.height && col < size.width)
        {
            auto const point = PixelToPoint({col, row}, size, topLeft, bottomRight);
            auto const value = GenerateTricornPixel(point, iterations, logIterations, radiusSquared);
            image[(row * size.width + col) * CHANNELS + 0] = value;
            image[(row * size.width + col) * CHANNELS + 1] = value;
            image[(row * size.width + col) * CHANNELS + 2] = value;
        }
    }

    __device__ auto GenerateTricornPixel(Point const startPoint, std::size_t const iterations, float const logIterations, float const radiusSquared) -> std::uint8_t
    {
        Point point{0.0, 0.0};

        for (std::size_t iteration = 0; iteration < iterations; ++iteration)
        {
            if (point.norm() > radiusSquared)
            {
                return static_cast<std::uint8_t>(MAX_COLOR * logf(iteration + 1) / logIterations);
            }

            auto const conjugate = point.conj();
            point = conjugate * conjugate + startPoint;
        }

        return 0;
    }

    __global__ static auto CosineKernel(std::uint8_t * const image, Size const size, std::size_t const iterations, float const logIterations, Point const topLeft,
    Point const bottomRight, float const radiusSquared) -> void
    {
        auto const row{(blockIdx.y * blockDim.y) + threadIdx.y};
        auto const col{(blockIdx.x * blockDim.x) + threadIdx.x};

        if (row < size.height && col < size.width)
        {
            auto const point = PixelToPoint({col, row}, size, topLeft, bottomRight);
            auto const value = GenerateCosinePixel(point, iterations, logIterations, radiusSquared);
            image[(row * size.width + col) * CHANNELS + 0] = value;
            image[(row * size.width + col) * CHANNELS + 1] = value;
            image[(row * size.width + col) * CHANNELS + 2] = value;
        }
    }

    __device__ auto GenerateCosinePixel(Point const startPoint, std::size_t const iterations, float const logIterations, float const radiusSquared) -> std::uint8_t
    {
        Point point{0.0, 0.0};

        for (std::size_t iteration = 0; iteration < iterations; ++iteration)
        {
            if (point.norm() > radiusSquared)
            {
                return static_cast<std::uint8_t>(MAX_COLOR * logf(iteration + 1) / logIterations);
            }

            point = point.cos() + startPoint;
        }

        return 0;
    }
} // namespace Fractal
